#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>

#define A(i,j) A[(i) + (j)*lda]
#define B(i,j) B[(i) + (j)*ldb]
#define C(i,j) C[(i) + (j)*ldc]
#define sa7(i,j) sa7[((j)<<6) + (i)]
#define sb7(i,j) sb7[((j)<<6) + (i)]
#define MS_7 64
#define NS_7 64
#define KS_7 16
#define M 8192
#define N 8192
#define K 1024
// 分块大小
#define BM 64
#define BN 64
#define BK 16
// #define A(i,j) A[(i) + (j)*lda]
// #define B(i,j) B[(i) + (j)*ldb]
// #define C(i,j) C[(i) + (j)*ldc]
#define IDX2C(i, j, ld) ((j) * (ld) + (i)) // columb-major
//v1 += v2 * s3, vector scaling
#define vscal(v1, v2, s3)\
    v1.x+=v2.x*s3;\
    v1.y+=v2.y*s3;\
    v1.z+=v2.z*s3;\
    v1.w+=v2.w*s3;
//v1 = alpha * v2 + beta * v3, simd fma
#define simd_axpby(v1, alpha, v2, beta, v3)\
    v1.x=alpha*v2.x+beta*v3.x;\
    v1.y=alpha*v2.y+beta*v3.y;\
    v1.z=alpha*v2.z+beta*v3.z;\
    v1.w=alpha*v2.w+beta*v3.w;
#define vload(v1,addr)\
    v1 = *((float4 *)(addr));
#define vstore(addr,v1)\
    *((float4 *)(addr)) = v1;
#include<stdio.h>
#include<stdlib.h>
#define A(i,j) A[(i) + (j)*lda]
#define B(i,j) B[(i) + (j)*ldb]
#define ptr_A(i,j) ptr_A[(i) + (j)*lda]
#define ptr_B(i,j) ptr_B[(i) + (j)*ldb]
#define C(i,j) C[(i) + (j)*ldc]
#define sa10(i,j) sa10[((j)<<7) + (i)]
#define sb10(i,j) sb10[((j)<<7) + (i)]
#define MS_10 128
#define NS_10 128
#define KS_10 8
//v1 += v2 * s3, vector scaling
#define vscal(v1, v2, s3)\
    v1.x+=v2.x*s3;\
    v1.y+=v2.y*s3;\
    v1.z+=v2.z*s3;\
    v1.w+=v2.w*s3;
//v1 = alpha * v2 + beta * v3, simd fma
#define simd_axpby(v1, alpha, v2, beta, v3)\
    v1.x=alpha*v2.x+beta*v3.x;\
    v1.y=alpha*v2.y+beta*v3.y;\
    v1.z=alpha*v2.z+beta*v3.z;\
    v1.w=alpha*v2.w+beta*v3.w;
#define vload(v1,addr)\
    v1 = *((float4 *)(addr));
#define vstore(addr,v1)\
    *((float4 *)(addr)) = v1;
// cache blocking version, without register-level data re-use
// with memory coelascing on shared memory
// more workloads per thread. 8x8 micro kernel.
// adopt vetorized load/store
__global__ void naive_matmul(const int m,const int n,const int k,const float alpha, const float *A, const float *B, const float beta, float* C)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    A = &A[IDX2C(bx<<5,0,m)]; // blockdim(32,32)
    B = &B[IDX2C(0,by<<5,k)];
    C = &C[IDX2C(bx<<5,by<<5,m)];
    float sum = 0.0;
    for (int i = 0; i < k; i++){
        sum += A[IDX2C(tx,i,m)] * B[IDX2C(i,ty,k)];
    }
    C[IDX2C(tx,ty,m)] = alpha * sum + beta * C[IDX2C(tx,ty,m)];
}
// cache blocking version, without register-level data re-use
// with memory coelascing on shared memory
// more workloads per thread. 4x4 micro kernel.
// adopt vetorized load/store
// __global__  __launch_bounds__(256)
__global__ void mysgemm_v7(int m, int n, int k, float alpha, float* A, float* B, float beta, float* C)
{
    int lda = M, ldb = K, ldc = M;
    int tx = threadIdx.x;
    int bx = blockIdx.x, by = blockIdx.y;
    int row_a = (tx&15)<<2, col_a = tx>>4;
    int row_b = (tx&3)<<2, col_b = tx>>2;
    int col_c = col_a<<2;
    int lda16 = lda<<4;
    A = &A((bx<<6),0); // 一个block256线程 解决64*64个元素
    B = &B(0,(by<<6));
    C = &C((bx<<6),(by<<6));//the TB size is 64.
    __shared__ float sa7[1024];
    __shared__ float sb7[1024];
    float4 Av, Bv, Cv[4], Cres[4];
    memset(Cres, 0, sizeof(Cres)); //
    for (int k_count = 0; k_count<K; k_count+=KS_7){
        vload(Av, &A[IDX2C(row_a, col_a, lda)])
        vload(Bv, &B(row_b, col_b))
        ((float4 *)sa7)[tx] = Av;
        sb7(col_b,row_b)=Bv.x;
        sb7(col_b,row_b+1)=Bv.y;
        sb7(col_b,row_b+2)=Bv.z;
        sb7(col_b,row_b+3)=Bv.w;
        A+=lda16;B+=16;
        __syncthreads();
        #pragma unroll
        for (int inner_k_count=0;inner_k_count<KS_7;inner_k_count++){
            vload(Av, &sa7[IDX2C(row_a, inner_k_count, BM)])
            vload(Bv, &sb7[IDX2C(col_c, inner_k_count, BM)])
            // vload(Av, &sa7(row_a,inner_k_count))
            // vload(Bv, &sb7(col_c,inner_k_count))
            vscal(Cres[0], Av, Bv.x)
            vscal(Cres[1], Av, Bv.y)
            vscal(Cres[2], Av, Bv.z)
            vscal(Cres[3], Av, Bv.w)
        }
        __syncthreads();
    }
    vload(Cv[0], &C[IDX2C(row_a,col_c,m)])
    vload(Cv[1], &C[IDX2C(row_a,col_c+1,m)])
    vload(Cv[2], &C[IDX2C(row_a,col_c+2,m)])
    vload(Cv[3], &C[IDX2C(row_a,col_c+3,m)]) // 向量化读
    simd_axpby(Cres[0],alpha,Cres[0],beta,Cv[0])
    simd_axpby(Cres[1],alpha,Cres[1],beta,Cv[1])
    simd_axpby(Cres[2],alpha,Cres[2],beta,Cv[2])
    simd_axpby(Cres[3],alpha,Cres[3],beta,Cv[3])

    vstore(&C[IDX2C(row_a,col_c, m)], Cres[0])
    vstore(&C[IDX2C(row_a,col_c + 1, m)], Cres[1])
    vstore(&C[IDX2C(row_a,col_c + 2, m)], Cres[2])
    vstore(&C[IDX2C(row_a,col_c + 3, m)], Cres[3])  // 向量化写
}

// __global__  __launch_bounds__(256)
__global__ void mysgemm_v10(int m, int n, int k, float alpha, float* A, float* B, float beta, float* C){
    int lda = M, ldb = K, ldc = M;
    int tx = threadIdx.x;
    int bx = blockIdx.x, by = blockIdx.y;
    int warp_id = tx>>5;
    int lane_id = tx&31;
    int warp_row = warp_id & 3, warp_col = warp_id >> 2;
    int row_w = lane_id&3, col_w = lane_id>>2;
    int row_b = (tx&1)<<2, col_b = tx>>1;
    int lda8 = lda<<3;
    int row_c = (warp_row<<5) + (row_w<<3), col_c = (warp_col<<6) + (col_w<<3);
    int row_a = (tx&31)<<2, col_a = tx>>5;
    int K_upper = K>>3;
    A = &A((bx<<7),0);
    B = &B(0,(by<<7));
    C = &C((bx<<7),(by<<7));//the TB size is 128.
    __shared__ float sa10[1024];
    __shared__ float sb10[1024];
    float4 Av1[2], Av2[2], Bv1[2], Bv2[2], Cv[16], Cres[16];
    float4 pref_Av, pref_Bv;
    float* ptr_A, *ptr_B;
    memset(Cres, 0, sizeof(Cres));//clear registers
    vload(pref_Av, &A(row_a,col_a))
    vload(pref_Bv, &B(row_b,col_b))
    ((float4 *)sa10)[tx] = pref_Av;
    sb10(col_b,row_b)=pref_Bv.x;
    sb10(col_b,row_b+1)=pref_Bv.y;
    sb10(col_b,row_b+2)=pref_Bv.z;
    sb10(col_b,row_b+3)=pref_Bv.w;
    __syncthreads();
    vload(Av1[0], &sa10(row_c,0))
    vload(Av2[0], &sa10(row_c+4,0))
    vload(Bv1[0], &sb10(col_c,0))
    vload(Bv2[0], &sb10(col_c+4,0))
    for (int k_count = 0; k_count<K_upper; k_count++){
        /*packing A and B into shared memory*/
        int inc = (k_count+1)%K_upper;
        ptr_A = A + inc * lda8;
        ptr_B = B + inc * 8;
        vload(pref_Av, &ptr_A(row_a,col_a))
        vload(pref_Bv, &ptr_B(row_b,col_b))
        #pragma unroll
        for (int inner_k_count=0;inner_k_count<KS_10;inner_k_count++){
            int next_inner_k_count = (inner_k_count+1)&7;
            vload(Av1[(inner_k_count+1)&1], &sa10(row_c,next_inner_k_count))
            vload(Av2[(inner_k_count+1)&1], &sa10(row_c+4,next_inner_k_count))
            vload(Bv1[(inner_k_count+1)&1], &sb10(col_c,next_inner_k_count))
            vload(Bv2[(inner_k_count+1)&1], &sb10(col_c+4,next_inner_k_count))
            vscal(Cres[0], Av1[(inner_k_count)&1], Bv1[(inner_k_count)&1].x)
            vscal(Cres[1], Av2[(inner_k_count)&1], Bv1[(inner_k_count)&1].x)
            vscal(Cres[2], Av1[(inner_k_count)&1], Bv1[(inner_k_count)&1].y)
            vscal(Cres[3], Av2[(inner_k_count)&1], Bv1[(inner_k_count)&1].y)
            vscal(Cres[4], Av1[(inner_k_count)&1], Bv1[(inner_k_count)&1].z)
            vscal(Cres[5], Av2[(inner_k_count)&1], Bv1[(inner_k_count)&1].z)
            vscal(Cres[6], Av1[(inner_k_count)&1], Bv1[(inner_k_count)&1].w)
            vscal(Cres[7], Av2[(inner_k_count)&1], Bv1[(inner_k_count)&1].w)
            vscal(Cres[8], Av1[(inner_k_count)&1], Bv2[(inner_k_count)&1].x)
            vscal(Cres[9], Av2[(inner_k_count)&1], Bv2[(inner_k_count)&1].x)
            vscal(Cres[10], Av1[(inner_k_count)&1], Bv2[(inner_k_count)&1].y)
            vscal(Cres[11], Av2[(inner_k_count)&1], Bv2[(inner_k_count)&1].y)
            vscal(Cres[12], Av1[(inner_k_count)&1], Bv2[(inner_k_count)&1].z)
            vscal(Cres[13], Av2[(inner_k_count)&1], Bv2[(inner_k_count)&1].z)
            vscal(Cres[14], Av1[(inner_k_count)&1], Bv2[(inner_k_count)&1].w)
            vscal(Cres[15], Av2[(inner_k_count)&1], Bv2[(inner_k_count)&1].w)
        }
        __syncthreads();
        ((float4 *)sa10)[tx] = pref_Av;
        sb10(col_b,row_b)=pref_Bv.x;
        sb10(col_b,row_b+1)=pref_Bv.y;
        sb10(col_b,row_b+2)=pref_Bv.z;
        sb10(col_b,row_b+3)=pref_Bv.w;
        __syncthreads();
        vload(Av1[0], &sa10(row_c,0))
        vload(Av2[0], &sa10(row_c+4,0))
        vload(Bv1[0], &sb10(col_c,0))
        vload(Bv2[0], &sb10(col_c+4,0))
    }
    vload(Cv[0], &C(row_c,col_c))
    vload(Cv[1], &C(row_c+4,col_c))
    vload(Cv[2], &C(row_c,col_c+1))
    vload(Cv[3], &C(row_c+4,col_c+1))
    vload(Cv[4], &C(row_c,col_c+2))
    vload(Cv[5], &C(row_c+4,col_c+2))
    vload(Cv[6], &C(row_c,col_c+3))
    vload(Cv[7], &C(row_c+4,col_c+3))
    vload(Cv[8], &C(row_c,col_c+4))
    vload(Cv[9], &C(row_c+4,col_c+4))
    vload(Cv[10], &C(row_c,col_c+5))
    vload(Cv[11], &C(row_c+4,col_c+5))
    vload(Cv[12], &C(row_c,col_c+6))
    vload(Cv[13], &C(row_c+4,col_c+6))
    vload(Cv[14], &C(row_c,col_c+7))
    vload(Cv[15], &C(row_c+4,col_c+7))
    
    simd_axpby(Cres[0],alpha,Cres[0],beta,Cv[0])
    simd_axpby(Cres[1],alpha,Cres[1],beta,Cv[1])
    simd_axpby(Cres[2],alpha,Cres[2],beta,Cv[2])
    simd_axpby(Cres[3],alpha,Cres[3],beta,Cv[3])

    simd_axpby(Cres[4],alpha,Cres[4],beta,Cv[4])
    simd_axpby(Cres[5],alpha,Cres[5],beta,Cv[5])
    simd_axpby(Cres[6],alpha,Cres[6],beta,Cv[6])
    simd_axpby(Cres[7],alpha,Cres[7],beta,Cv[7])

    simd_axpby(Cres[8],alpha,Cres[8],beta,Cv[8])
    simd_axpby(Cres[9],alpha,Cres[9],beta,Cv[9])
    simd_axpby(Cres[10],alpha,Cres[10],beta,Cv[10])
    simd_axpby(Cres[11],alpha,Cres[11],beta,Cv[11])

    simd_axpby(Cres[12],alpha,Cres[12],beta,Cv[12])
    simd_axpby(Cres[13],alpha,Cres[13],beta,Cv[13])
    simd_axpby(Cres[14],alpha,Cres[14],beta,Cv[14])
    simd_axpby(Cres[15],alpha,Cres[15],beta,Cv[15])

    vstore(&C(row_c,col_c), Cres[0])
    vstore(&C(row_c+4,col_c), Cres[1])
    vstore(&C(row_c,col_c+1), Cres[2])
    vstore(&C(row_c+4,col_c+1), Cres[3])
    vstore(&C(row_c,col_c+2), Cres[4])
    vstore(&C(row_c+4,col_c+2), Cres[5])
    vstore(&C(row_c,col_c+3), Cres[6])
    vstore(&C(row_c+4,col_c+3), Cres[7])
    vstore(&C(row_c,col_c+4), Cres[8])
    vstore(&C(row_c+4,col_c+4), Cres[9])
    vstore(&C(row_c,col_c+5), Cres[10])
    vstore(&C(row_c+4,col_c+5), Cres[11])
    vstore(&C(row_c,col_c+6), Cres[12])
    vstore(&C(row_c+4,col_c+6), Cres[13])
    vstore(&C(row_c,col_c+7), Cres[14])
    vstore(&C(row_c+4,col_c+7), Cres[15])
}
void gpuSgemm(int m, int n, int k, const float *alpha, 
    const float *A, const float *B, const float *beta, float *C) {
        int blocksize = 256;
        // int GridSize = ceil(sqrt((N+bs-1.) / bs));
        // int GridSize = ceil((M*N+blocksize-1.) / blocksize);
        int gridx = floor(M/BM);
        int gridy = floor(N/BN);
        dim3 Grid(gridx, gridy); //
        dim3 Block(256); // 32 * 32 = 1024  
        //malloc on device
        float *devPtrA, *devPtrB, *devPtrC,*devPtrD;
        hipMalloc((void**)&devPtrA, sizeof(float) * m * k);
        hipMalloc((void**)&devPtrB, sizeof(float) * k * n);
        hipMalloc((void**)&devPtrC, sizeof(float) * m * n);
        hipMalloc((void**)&devPtrD, sizeof(float) * m * n);
        //copy A and B to device
        hipMemcpy(devPtrA, A, m * k * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(devPtrB, B, k * n * sizeof(float), hipMemcpyHostToDevice);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
// ------------------------------------------------------------------------------------
        mysgemm_v10<<<Grid,Block>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrC);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
    
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("gpu with gemm_shared_v2 kernel time:%f ms\n",milliseconds);
        float* matrix_out_cpu=(float*)malloc(sizeof(float) * M * N);
        float* matrix_out_gpu=(float*)malloc(sizeof(float) * M * N);
        hipMemcpy(matrix_out_cpu, devPtrC, m * n * sizeof(float), hipMemcpyDeviceToHost);
        dim3 Grid_n(M/32, N/32); //
        dim3 Block_n(32,32); // 32 * 32 = 1024  
        naive_matmul<<<Grid_n,Block_n>>>(m,n,k,*alpha,devPtrA,devPtrB,*beta,devPtrD);
        hipMemcpy(matrix_out_gpu, devPtrD, m * n * sizeof(float), hipMemcpyDeviceToHost);

        float EPSILON = 0.01;
        // check result                                             
        printf("check\n");
        for (int i = 0; i < M * N; ++i) {
            float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
                / matrix_out_gpu[i];
            if (error < -EPSILON || error > EPSILON)
                printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
                    error);
        }
        printf("right\n");

        //release memory on device
        hipFree(devPtrA);
        hipFree(devPtrB);
        hipFree(devPtrC);
        hipFree(devPtrD);
        free(matrix_out_cpu);
        free(matrix_out_gpu);
}

int main(){
    float rand_min = -10.0, rand_max = 10.0, rand_num = 0.0;

    float* matrix_in1 = (float*)malloc(sizeof(float) * M * K);
    float* matrix_in2 = (float*)malloc(sizeof(float) * K * N);
    float* matrix_out_cpu = (float*)malloc(sizeof(float) * M * N);
    float* matrix_out_gpu = (float*)malloc(sizeof(float) * M * N);

    for (int i = 0; i< M * K; i++){
        rand_num = (float)rand() / RAND_MAX; // RAND_MAX = 32767
        matrix_in1[i] = rand_min + rand_num * (rand_max - rand_min);
    }
    for (int i = 0; i < K * N; ++i) {
        rand_num = (float)rand()/RAND_MAX;
        matrix_in2[i] = rand_min + rand_num * (rand_max - rand_min);
    }

    clock_t start, stop;
    float a = 1.0, b = 0.0;
    double duration;
    
    // // record cpu execution time
    // start=clock();
    // cpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_cpu);
    // stop=clock();
    // duration=(double)(stop-start)/CLOCKS_PER_SEC;
    // printf("cpu time:%f\n",duration);

    ///////////////////////////////////////////////////////////////////////////////////
    gpuSgemm(M, N, K, &a, matrix_in1, matrix_in2, &b, matrix_out_gpu);
  
    // float EPSILON = 0.1;
    // // check result                                             
    // printf("check\n");
    // for (int i = 0; i < M * N; ++i) {
    //     float error = (matrix_out_cpu[i] - matrix_out_gpu[i]) 
    //         / matrix_out_gpu[i];
    //     if (error < -EPSILON || error > EPSILON)
    //         printf("wrong, %f, %f, %f\n", matrix_out_cpu[i], matrix_out_gpu[i], 
    //             error);
    // }
    // printf("right\n");

    //release memory on host
    free(matrix_in1);
    free(matrix_in2);
    free(matrix_out_cpu);
    free(matrix_out_gpu);

    return 0;
}